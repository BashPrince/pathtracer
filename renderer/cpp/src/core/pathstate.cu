#include "pathstate.hpp"
#include "../util/util.hpp"

PathstateSoA::PathstateSoA()
    :
    currentIsect(nullptr),
    extensionIsect(nullptr),
    foundExtensionIsect(nullptr),
    isActive(nullptr),
    pdf(nullptr),
    lightPdf(nullptr),
    bounces(nullptr),
    beta(nullptr),
    L(nullptr),
    wi_L(nullptr),
    Lemit(nullptr),
    Lsample(nullptr),
    LsampleOccluded(nullptr),
    isLightSample(nullptr),
    filmIndex(nullptr),
    material(nullptr),
    f(nullptr),
    light_f(nullptr),
    diffuseBounce(nullptr)
{}

void PathstateSoA::freeArrays() {
        if(currentIsect) {
            hipFree(currentIsect);
            waitAndCheckError("PathstateSoA::freeArrays(currentIsect)");
            currentIsect = nullptr;
        }
        if(extensionIsect) {
            hipFree(extensionIsect);
            waitAndCheckError("PathstateSoA::freeArrays(extensionIsect)");
            extensionIsect = nullptr;
        }
        if(foundExtensionIsect) {
            hipFree(foundExtensionIsect);
            waitAndCheckError("PathstateSoA::freeArrays(foundExtensionIsect)");
            foundExtensionIsect = nullptr;
        }
        if(isActive) {
            hipFree(isActive);
            waitAndCheckError("PathstateSoA::freeArrays(isActive)");
            isActive = nullptr;
        }
        if(pdf) {
            hipFree(pdf);
            waitAndCheckError("PathstateSoA::freeArrays(pdf)");
            pdf = nullptr;
        }
        if(lightPdf) {
            hipFree(lightPdf);
            waitAndCheckError("PathstateSoA::freeArrays(lightPdf)");
            lightPdf = nullptr;
        }
        if(bounces) {
            hipFree(bounces);
            waitAndCheckError("PathstateSoA::freeArrays(bounces)");
            bounces = nullptr;
        }
        if(beta) {
            hipFree(beta);
            waitAndCheckError("PathstateSoA::freeArrays(beta)");
            beta = nullptr;
        }
        if(L) {
            hipFree(L);
            waitAndCheckError("PathstateSoA::freeArrays(L)");
            L = nullptr;
        }
        if(wi_L) {
            hipFree(wi_L);
            waitAndCheckError("PathstateSoA::freeArrays(wi_L)");
            wi_L = nullptr;
        }
        if(Lemit) {
            hipFree(Lemit);
            waitAndCheckError("PathstateSoA::freeArrays(Lemit)");
            Lemit = nullptr;
        }
        if(filmIndex) {
            hipFree(filmIndex);
            waitAndCheckError("PathstateSoA::freeArrays(filmIndex)");
            filmIndex = nullptr;
        }
        if(material) {
            hipFree(material);
            waitAndCheckError("PathstateSoA::freeArrays(material)");
            material = nullptr;
        }
        if(f) {
            hipFree(f);
            waitAndCheckError("PathstateSoA::freeArrays(f)");
            f = nullptr;
        }
        if(light_f) {
            hipFree(light_f);
            waitAndCheckError("PathstateSoA::freeArrays(light_f)");
            light_f = nullptr;
        }
        if(Lsample) {
            hipFree(Lsample);
            waitAndCheckError("PathstateSoA::freeArrays(Lsample)");
            Lsample = nullptr;
        }
        if(LsampleOccluded) {
            hipFree(LsampleOccluded);
            waitAndCheckError("PathstateSoA::freeArrays(LsampleOccluded)");
            LsampleOccluded = nullptr;
        }
        if(isLightSample) {
            hipFree(isLightSample);
            waitAndCheckError("PathstateSoA::freeArrays(isLightSample)");
            isLightSample = nullptr;
        }
        if(diffuseBounce) {
            hipFree(diffuseBounce);
            waitAndCheckError("PathstateSoA::freeArrays(diffuseBounce)");
            diffuseBounce = nullptr;
        }
    }